#include "hip/hip_runtime.h"
#include "virtual_memory.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
__device__ int vm_LRU(VirtualMemory *vm);
__device__ inline void save2memory(VirtualMemory *vm, int index, uchar value);
__device__ void swap_out(VirtualMemory *vm, int page_num, int frame_num);
__device__ void swap_in(VirtualMemory *vm, int mem_num, int origin_num, int disk_num);

//This variable save the LRU index.
__device__ static int index;

__device__ void init_invert_page_table(VirtualMemory *vm) {

    for (int i = 0; i < vm->PAGE_ENTRIES; i++) {

        /* Use the first number as the invalid-valid checker
         * If any data is saved to the memory and changed the page table,
         * The MSB will not be 1 and become valid.
         */
        // 0-1023: Record the frame number in the physical memory and
        vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
        // Page Entries: 1024 entries in the memory and the page table.
        // 1024-2047: Record the frame number in the secondary memory
        vm->invert_page_table[i + vm->PAGE_ENTRIES] = 0x80000000;
        // 2048-3071: Count the used numbers to manipulate LRU algorithm.
        vm -> invert_page_table[i + 2 * vm->PAGE_ENTRIES] = 0;
        /* 3072-4096: Modified Bit: 0 -> Not Modified, can access it directly from the memory
         * 1 -> Modified, should raise the page fault and let the OS to get it from the disk.
         */
        vm -> invert_page_table[i + 3 * vm->PAGE_ENTRIES] = 0;
    }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
    // init variables
    vm->buffer = buffer;
    vm->storage = storage;
    vm->invert_page_table = invert_page_table;
    vm->pagefault_num_ptr = pagefault_num_ptr;

    // init constants
    vm->PAGESIZE = PAGESIZE;
    vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
    vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
    vm->STORAGE_SIZE = STORAGE_SIZE;
    vm->PAGE_ENTRIES = PAGE_ENTRIES;

    // before first vm_write or vm_read
    // Initialize the page table.
    init_invert_page_table(vm);
}



__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
    /* Complete vm_write function to write value into data buffer */
    u32 page_number = addr / 32;
    // This remainder's value is the first goal of the memory paging.
    u32 remainder = page_number % 1024;
//    printf("%d",int(remainder));
    // Only raise the page fault and do the swap out at the first byte of the page.
    if(addr % 32 == 0){
        // Two situations:
        // 1. Empty Page Table, Empty memory
        if(vm->invert_page_table[remainder] == 0x80000000){
            *(vm->pagefault_num_ptr) += 1;
            // Update the page table:
            vm->invert_page_table[remainder] = remainder;
            vm->invert_page_table[remainder + vm->PAGE_ENTRIES] = page_number;
            //vm->invert_page_table[remainder + 2 * vm->PAGE_ENTRIES] = addr;
            vm->invert_page_table[remainder + 2 * vm->PAGE_ENTRIES] = page_number;

            save2memory(vm, 32 * vm->invert_page_table[remainder], value);
            index = remainder;

        }
        else{
            // 2. The page table (memory) is full.
            *(vm->pagefault_num_ptr) += 1;
            // Use LRU to find the position we want to replace.
//            printf("PAGE:%d", page_number);

            index = vm_LRU(vm);
            // Change the page number in the page table.
            // Do the swap out.
//            printf("INDEX: %d", index);

            swap_out(vm, vm->invert_page_table[index], vm->invert_page_table[index + vm->PAGE_ENTRIES]);
            // Then, update the page table.
            vm->invert_page_table[index] = remainder;
            vm->invert_page_table[index + vm->PAGE_ENTRIES] = page_number;
            vm->invert_page_table[index + 2 * vm->PAGE_ENTRIES] = page_number;
            // Set the modified bit to show that it has been modified.
            vm->invert_page_table[index + 3 * vm->PAGE_ENTRIES] = 1;
            save2memory(vm, 32 * vm->invert_page_table[index], value);
        }

    }
    else{
        // Deal with the bytes in the page:
        save2memory(vm, (32 * vm->invert_page_table[index]) + (addr % 32), value);
    }

    // For the other
    // Overwrite, 如果已存在自己到最常用，其它的往后掉。
    // New: 如果不存在，则放到最后一位，常用度最高。
    // 被访问：放到最后一位，常用度最高。
}

// This function will read data from the input buffer.
__device__ uchar vm_read(VirtualMemory *vm, u32 addr, int & max_count) {
    u32 page_number = addr / 32;
    u32 remainder = page_number % 1024;

    for (int i = 0; i < vm->PAGE_ENTRIES; ++i) {
        // Situation 1: Get the data from the memory directly.
        if (vm->invert_page_table[i + vm->PAGE_ENTRIES] == page_number) {
            index = i;
            printf("INDEX: %d\n", index);
            vm->invert_page_table[remainder + 2 * vm->PAGE_ENTRIES] = ++max_count;
            return vm->buffer[32 * index + (addr % 32)];
        }
    }


    //Situation 2: The data we need is not in the memory. We should get the result from the disk by swapping in.
    printf("SWAP");
    index = vm_LRU(vm);
    printf("INDEX: %d", index);
    //Update the page fault information
    (*vm->pagefault_num_ptr) += 1;

    //Swap in the data
    int origin_number = vm->invert_page_table[index + vm->PAGE_ENTRIES];
    //Save data to the disk[origin_number], and load disk[page_number] to mem[index]
    swap_in(vm, vm->invert_page_table[index],page_number, origin_number);
    // Update the page table
    vm->invert_page_table[index] = index;
    vm->invert_page_table[index + vm->PAGE_ENTRIES] = page_number;
    vm->invert_page_table[index + 2 * vm->PAGE_ENTRIES] = max_count + 1;
    return vm->buffer[32 * vm->invert_page_table[index] + (addr % 32)];

    // Look through all over the page table to find whether the data we need is in the memory.

}


__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size, int & max_count) {
    /* Complete snapshot function togther with vm_read to load elements from data
     * to result buffer */
    for (int i = 0; i < input_size; ++i){
        int value = vm_read(vm, i, max_count);
        results[offset + i] = value;
    }
}

// This function return the appropriate index to replace.
__device__ int vm_LRU(VirtualMemory *vm){
    // This two integers help to find the least used entity.
    int min_index = 0;
    int min = INT_MAX;
    int temp = 0;
    for (int i = 0; i < vm->PAGE_ENTRIES; ++i) {
        temp = vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES];
        if (temp == 0) {
            return i;
        }
        if (temp <= min){
            min = temp;
            min_index = i;
        }
    }
    return min_index;
}

__device__ inline void save2memory(VirtualMemory *vm, int index, uchar value){
    vm->buffer[index] = value;
}

__device__ void swap_out(VirtualMemory *vm, int page_num, int frame_num){
    for (int i = 0; i < vm->PAGESIZE; ++i){
        vm->storage[32 * frame_num + i] = vm->buffer[32 * page_num + i];
        vm->buffer[32 * page_num + i] = NULL;
    }

}
__device__  void swap_in(VirtualMemory *vm, int mem_num, int disk_num, int origin_num){
    for (int i = 0; i < vm->PAGESIZE; ++i){
        vm->storage[32 * origin_num + i] = vm->buffer[32 * mem_num + i];
    }
    for (int i = 0; i < vm->PAGESIZE; ++i) {
        vm->buffer[32 * mem_num + i] = vm->storage[32 * disk_num + i];
    }

}