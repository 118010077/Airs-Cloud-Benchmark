#include "virtual_memory.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size) {
    int max_count = 0;
    for (int i = 0; i < input_size; i++){
        vm_write(vm, i, input[i]);
    }

//    vm_read(vm, input_size);
    for(int i = 0; i < vm->PAGE_ENTRIES; ++ i){
        if(vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] > max_count){
            max_count = vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES];
        }
    }

    for (int i = input_size - 1; i >= input_size - 32769; i--){
        int value = vm_read(vm, i, max_count);
    }

    // Read the data to the result buffer, and this buffer is exactly the output buffer.
    vm_snapshot(vm, results, 0, input_size, max_count);

}
