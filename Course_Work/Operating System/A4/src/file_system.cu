#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BYTE_TO_BINARY_PATTERN "0b%c%c%c%c%c%c%c\n"
#define BYTE_TO_BINARY(byte)  \
  (byte & 0x80 ? '1' : '0'), \
  (byte & 0x40 ? '1' : '0'), \
  (byte & 0x20 ? '1' : '0'), \
  (byte & 0x10 ? '1' : '0'), \
  (byte & 0x08 ? '1' : '0'), \
  (byte & 0x04 ? '1' : '0'), \
  (byte & 0x02 ? '1' : '0'), \
  (byte & 0x01 ? '1' : '0')


__device__ __managed__ u32 gtime = 0;
//We have a 128 bytes TempArray for sorting.
__device__ __managed__ uchar tempArray[128];
__device__ void fs_gsys(FileSystem *fs, int op, char *s);

__device__ void init_volume(FileSystem *fs) {
    for (int i = 0; i < fs->SUPERBLOCK_SIZE; ++i){
        fs->volume[i] = 0b11111111;
    }
    for (int i = fs->SUPERBLOCK_SIZE; i < fs->STORAGE_SIZE;++i){
        fs->volume[i] = 0;
    }

}

// This function translate 4 u8 chars to u32.
__device__ inline u32 u8tou32(FileSystem * fs, u32 i){
    // 1st byte + 2nd byte * 2^8 + 3rd byte * 2^16 + 4th byte * 2^24
    return u32(fs->volume[i]) + u32(fs->volume[i + 1] * 256) + u32(fs->volume[i + 2] * 65536)
    + u32(fs->volume[i + 3] * 16777216);
}

// This function translate a u32 number to 4 u8 chars.
__device__ void u32tou8(FileSystem * fs, u32 index, u32 value){
    fs->volume[index] = value & 0x000000FFu;
    fs->volume[index+1] = (value & 0x0000FF00u) >> 8u;
    fs->volume[index+2] = (value & 0x00FF0000u) >> 16u;
    fs->volume[index+3] = (value & 0xFF000000u) >> 24u;
}

// This function update the super block (bit map).
//Start: First byte address of the file.
__device__ void updateSuperBlock(FileSystem * fs, u32 start, u32 size, int mode = 0){
    u32 block_index = (start - fs->FILE_BASE_ADDRESS)  / 32 + mode; // Block number of this file (0-1023)
    u32 super_index = block_index / 8; //Which entry this block belongs to in the super block.
    //Offset to find it in bit-map (0 - 7). 0: First bit, 7: Last bit
    u32 super_offset = block_index % 8;
    //Number of blocks this file need.
    u32 total_block;
    if(size % 32 == 0) total_block = size / 32;
    else total_block = (size / 32 + 1);
    //After the initialization, we update the Bit-Map
    //Mode == 1 -> Free the bit-map (modify 0 -> 1)
    if(mode == 1){
        for(u32 k = super_index + 1; k < fs->SUPERBLOCK_SIZE; ++k){
            fs->volume[k] = 0b11111111;
        }
        //Then change the information of super_index:
        switch(super_offset){
            case 0:
                fs->volume[super_index] = 0b11111111;
                break;
            case 1:
                fs->volume[super_index] = 0b01111111;
                break;
            case 2:
                fs->volume[super_index] = 0b00111111;
                break;
            case 3:
                fs->volume[super_index] = 0b00011111;
                break;
            case 4:
                fs->volume[super_index] = 0b00001111;
                break;
            case 5:
                fs->volume[super_index] = 0b00000111;
                break;
            case 6:
                fs->volume[super_index] = 0b00000011;
                break;
            case 7:
                fs->volume[super_index] = 0b00000001;
                break;
        }
        return;
    }
    //Mode == 0 -> Take up the bit-map (modify 1 -> 0)
    //Case 1: Just modify one super_index
    if(super_offset + total_block <= 8){
        switch(super_offset){
            case 0:
                switch(total_block){
                    case 1:
                        //Offset = 0 and only need 1 block. Change the first block to 0
                        fs->volume[super_index] = fs->volume[super_index] & 0b01111111u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00111111u;
                        break;
                    case 3:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00011111u;
                        break;
                    case 4:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00001111u;
                        break;
                    case 5:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00000111u;
                        break;
                    case 6:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00000011u;
                        break;
                    case 7:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00000001u;
                        break;
                    case 8:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00000000u;
                        break;
                }
                break;
            case 1:
                switch(total_block){
                    case 1:
                        //Offset = 0 and only need 1 block. Change the first block to 0
                        fs->volume[super_index] = fs->volume[super_index] & 0b10111111u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b10011111u;
                        break;
                    case 3:
                        fs->volume[super_index] = fs->volume[super_index] & 0b10001111u;
                        break;
                    case 4:
                        fs->volume[super_index] = fs->volume[super_index] & 0b10000111u;
                        break;
                    case 5:
                        fs->volume[super_index] = fs->volume[super_index] & 0b10000011u;
                        break;
                    case 6:
                        fs->volume[super_index] = fs->volume[super_index] & 0b10000001u;
                        break;
                    case 7:
                        fs->volume[super_index] = fs->volume[super_index] & 0b10000000u;
                        break;
                }
                break;
            case 2:
                switch(total_block){
                    case 1:
                        //Offset = 0 and only need 1 block. Change the first block to 0
                        fs->volume[super_index] = fs->volume[super_index] & 0b11011111u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11001111u;
                        break;
                    case 3:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11000111u;
                        break;
                    case 4:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11000011u;
                        break;
                    case 5:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11000001u;
                        break;
                    case 6:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11000000u;
                        break;
                }
                break;
            case 3:
                switch(total_block){
                    case 1:
                        //Offset = 0 and only need 1 block. Change the first block to 0
                        fs->volume[super_index] = fs->volume[super_index] & 0b11101111u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11100111u;
                        break;
                    case 3:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11100011u;
                        break;
                    case 4:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11100001u;
                        break;
                    case 5:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11100000u;
                        break;
                }
                break;
            case 4:
                switch(total_block){
                    case 1:
                        //Offset = 0 and only need 1 block. Change the first block to 0
                        fs->volume[super_index] = fs->volume[super_index] & 0b11110111u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11110011u;
                        break;
                    case 3:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11110001u;
                        break;
                    case 4:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11110000u;
                        break;
                }
                break;
            case 5:
                switch(total_block){
                    case 1:
                        //Offset = 0 and only need 1 block. Change the first block to 0
                        fs->volume[super_index] = fs->volume[super_index] & 0b11111011u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11111001u;
                        break;
                    case 3:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11111000u;
                        break;
                }
                break;
            case 6:
                switch(total_block){
                    case 1:
                        //Offset = 0 and only need 1 block. Change the first block to 0
                        fs->volume[super_index] = fs->volume[super_index] & 0b11111101u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b11111100u;
                        break;
                }
                break;
            case 7:
                fs->volume[super_index] = fs->volume[super_index] & 0b11111110u;
                break;
        }
    }
    else{
        int remain;
        remain = total_block - (8 - super_offset);
        //Case 2: Modify more super_index
        //Change the first index firstly.
        switch(super_offset){
            case 0:
                fs->volume[super_index] = fs->volume[super_index] & 0b00000000u;
                break;
            case 1:
                fs->volume[super_index] = fs->volume[super_index] & 0b10000000u;
                break;
            case 2:
                fs->volume[super_index] = fs->volume[super_index] & 0b11000000u;
                break;
            case 3:
                fs->volume[super_index] = fs->volume[super_index] & 0b11100000u;
                break;
            case 4:
                fs->volume[super_index] = fs->volume[super_index] & 0b11110000u;
                break;
            case 5:
                fs->volume[super_index] = fs->volume[super_index] & 0b11111000u;
                break;
            case 6:
                fs->volume[super_index] = fs->volume[super_index] & 0b11111100u;
                break;
            case 7:
                fs->volume[super_index] = fs->volume[super_index] & 0b11111110u;
                break;
        }
        //Change other bits.
        for(int i = 0; ;++i){
            if(remain / 8 == 0){
                switch(remain % 8){
                    case 1:
                        fs->volume[super_index] = fs->volume[super_index] & 0b01111111u;
                        break;
                    case 2:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00111111u;
                        break;
                    case 3:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00011111u;
                        break;
                    case 4:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00001111u;
                        break;
                    case 5:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00000111u;
                        break;
                    case 6:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00000011u;
                        break;
                    case 7:
                        fs->volume[super_index] = fs->volume[super_index] & 0b00000001u;
                        break;
                }
                break;
            }
            else{
                fs->volume[super_index + i] = 0b00000000u;
                remain /= 8;
            }
        }
    }

}

//Return a file_index if the file is found in FCB, otherwise return -1.
__device__ u32 find_file(FileSystem * fs, char * s){
    u32 file_index = -1;
    for (int i = fs->SUPERBLOCK_SIZE; i < fs->FILE_BASE_ADDRESS; i += 32){
        char subbuff[20];
        memcpy(subbuff, &(fs->volume[i]), 20);
        for(int j = 0; j < 20; ++j){
            if(subbuff[j] == s[j]){
                //If we find the file, we will reach \0
                if(s[j] == '\0'){
                    file_index = i;
                    return file_index;
                }
                else continue;
            }
            else{
                break;
            }
        }
    }
    return -1;
}

// Possibl Value for number: 0, 32, 64, ...
__device__ inline u32 get_index(FileSystem * fs, u32 number){
    return (number - fs->SUPERBLOCK_SIZE) / 32;
}
// This function check whether the index is already sorted out.
__device__ void update_used(FileSystem * fs, u32 index){
    u32 array_idx = index / 8; //array_idx: 0-127
    u32 offset = index % 8;  //offset: 0-7
    switch(offset){
        //Change the first bit to 0 (The first space is taken)
        case 0:
            tempArray[array_idx] = tempArray[array_idx] & 0b01111111u;
            break;
        case 1:
            tempArray[array_idx] = tempArray[array_idx] & 0b10111111u;
            break;
        case 2:
            tempArray[array_idx] = tempArray[array_idx] & 0b11011111u;
            break;
        case 3:
            tempArray[array_idx] = tempArray[array_idx] & 0b11101111u;
            break;
        case 4:
            tempArray[array_idx] = tempArray[array_idx] & 0b11110111u;
            break;
        case 5:
            tempArray[array_idx] = tempArray[array_idx] & 0b11111011u;
            break;
        case 6:
            tempArray[array_idx] = tempArray[array_idx] & 0b11111101u;
            break;
        case 7:
            tempArray[array_idx] = tempArray[array_idx] & 0b11111110u;
            break;
    }
}
// This function return whether the index is already be sorted out
__device__ bool is_used(FileSystem * fs, u32 index){
    u32 array_idx = index / 8; //array_idx: 0-127
    u32 offset = index % 8;  //offset: 0-7
    u32 check = tempArray[array_idx];
    switch(offset){
        //Check whether the first bit is 0 (taken)
        case 0:
            return (check >> 7u) & 0b00000001u;
        case 1:
            return (check >> 6u) & 0b00000001u;
        case 2:
            return (check >> 5u) & 0b00000001u;
        case 3:
            return (check >> 4u) & 0b00000001u;
        case 4:
            return (check >> 3u) & 0b00000001u;
        case 5:
            return (check >> 2u) & 0b00000001u;
        case 6:
            return (check >> 1u) & 0b00000001u;
        case 7:
            return check & 0b00000001u;
    }
}



//This function will compact all the following data to eliminate external fragmentation.
__device__ void compaction(FileSystem * fs, u32 empty_addr){
    //This addr is the address of the files that may be compacted.
    u32 compacted_addr;
    //Check every file that is saved "below" the removed file:
    for(int i = fs->SUPERBLOCK_SIZE + 20; i < fs->FILE_BASE_ADDRESS; i += 32){
        compacted_addr = u8tou32(fs, i);
        if(compacted_addr > empty_addr){
            //Move the file that will be compacted to the start address of the freed space
            u32 sizeofFile = u8tou32(fs, i+4);
            for(int k = 0; k < sizeofFile; ++k){
                fs->volume[empty_addr + k] = fs->volume[compacted_addr + k];
            }
            //Update the super block
            updateSuperBlock(fs, empty_addr, sizeofFile);
            //Update FCB
            //Update the compacted file's FCB by changing the start address to the freed space:
            u32tou8(fs, i, empty_addr);
            //Finally, we update the value of empty_addr to the file that we just compact:
            empty_addr = (empty_addr + sizeofFile - 1);
            //Adjust the value to fit into the block.
            if(empty_addr % 32 != 0){
                empty_addr = (empty_addr / 32 + 1) * 32;
            }
            else{
                continue;
            }
            break;
        }

    }
}
//This function will find a free block.
__device__ u32 get_Space(FileSystem * fs){
    int offset = 0;
    for(int i = 0; i < fs->SUPERBLOCK_SIZE; ++i){
        //Check from the larger scale (A cluster with 8 blocks)
        if(fs->volume[i] == 0) continue; //This 8 blocks is full.
        else if(fs->volume[i] == 255) {
            return i * 8;
        }

            //Then check which block is the last allocated space
        else{
            for(int k = 0x80; ; k /= 2){
                if(!(fs->volume[i] & k)){
                    ++offset;
                    continue;
                }
                else{
                    return i * 8 + offset;
                }
            }
        }
    }
}

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  // 0 ~~~ 4095: Super Block
  // 4096 ~~~ 36863: FCB Table
  // 36864 ~~~ 1085439; (1024KB 32768 Blocks): Contents of the file
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  init_volume(fs);
}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
    u32 file_index;
    //First of all, initialize the file pointer.
	u32 fp;
    char * t = s;
	//Look through the FCB table, and check whether we have the file:
	file_index = find_file(fs, s);
    // If file_index = -1, we did not find the file.
    if(file_index == -1){
        //If we want to write the file, we create a new file.
        if(op == G_WRITE){
            //Find an empty FCB entry first: By checking an enrty without file name
            for(int i = fs->SUPERBLOCK_SIZE; i < fs->FILE_BASE_ADDRESS; i += 32){
                if(fs->volume[i] == '\0'){
                    file_index = i;
                    break;
                }
            }
            //Copy the file name to the empty FCB entry.
            for(int i = 0; i < 20; ++i){
                if(s[i] == '\0') {
                    break;
                }
                fs->volume[file_index + i] = s[i];
            }

            // Create a new file after the last allocated block:
            // Find the offset(it is block not byte) first
            int free_block = get_Space(fs);
            // Then we update the FCB:
            // The "2nd parameter" of an FCB entry is the start address of the file
            u32 start_addr = 32 * free_block + fs->FILE_BASE_ADDRESS;
            u32tou8(fs, file_index+20, start_addr);
            return file_index;
        }
        else if (op == G_READ){
            printf("NOT FOUND THE FILE %s", s);
            return -1;
        }
    }

    return file_index;

}

//This function will read data from the disk to the output buffer.
__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
    //We find the file size and the file's start location first:
    u32 start_addr = u8tou32(fs, fp + 20);
    u32 file_size = u8tou32(fs, fp + 24);

    if(fp == -1) {
        printf("NO SUCH FILE IN FS");
        return;
    }
    else if (file_size < size){
//        printf("Start Output the Result");
        for(u32 i = 0; i < file_size; ++i){
//            printf("%c", fs->volume[start_addr + i]);
            output[i] = fs->volume[start_addr + i];
        }
    }
    else{
//        printf("Start Output the Result");
        for(u32 i = 0; i < size; ++i){
            output[i] = fs->volume[start_addr + i];
        }
    }

}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
    //Update the start address and calculate the old file_size.
    //Update the time:
    u32tou8(fs, fp+28, ++gtime);
    u32 start_addr = u8tou32(fs, fp + 20);
    u32 file_size = u8tou32(fs, fp + 24);
    //Two Situations:
    //1. The allocated space is larger than the new input.
    if(file_size > size){
        //In this situation, we just need to override the old file
        for(u32 i = 0; i < size; ++i){
            fs->volume[start_addr + i] = input[i];
        }
        //Update FCB:
        //We calculate how many blocks it will need first
        int block = size / 32;
        if(size % 32 != 0) ++block;
        u32tou8(fs, fp+24, size);
        //Modify the bit-map --- change all the following bits to 1
        updateSuperBlock(fs, start_addr + 32 * block, 0, 1);
        //This operation will compact the space and modify the bit map.
        compaction(fs, start_addr + block * 32);
    }
    //2. The allocated space is smaller than the new input(new file is created)
    else if (file_size < size){
        //Free the taken space first
        for(u32 i = 0; i < file_size; ++i){
            fs->volume[start_addr + i] = 0;
        }
        //Compact the disk:
        compaction(fs, start_addr);
        //Find a new free space to write the new file
        start_addr = 32 * get_Space(fs) + fs->FILE_BASE_ADDRESS;
        //Write Data
        for(u32 i = 0; i < size; ++i){
            fs->volume[start_addr + i] = input[i];
        }
        //Update FCB:
        //We calculate how many blocks it will need first
        int block = size / 32;
        if(size % 32 != 0) ++block;
        u32tou8(fs, fp + 24, size);
        //Then update the start address
        u32tou8(fs, fp + 20, start_addr);
        //Update Super Block
        updateSuperBlock(fs, start_addr, size);
    }
    //File_size = size;
    else{
        //We just need to update the content in the disk.
        for(u32 i = 0; i < size; ++i){
            fs->volume[start_addr + i] = input[i];
        }
    }
}

__device__ void fs_gsys(FileSystem *fs, int op)
{
    //Refresh the tempArray;
    for(int i = 0; i < 128; ++i){
        tempArray[i] = 0b11111111;
    }

    /*
     * This array has 128 entries,
     * we use each entry to represent 8 files just like a bit-map.
     */
	uchar * sorted_file = tempArray;
	u32 index = 0xffffffff; //Should be 0 - 1023
	u32 size_num; // Current size value
	u32 time_num; //Current time value
    u32 max_num = 0; //Max Value (Can be size or time)
    u32 max_index = 0; //Index with the Max Value
    u32 max_addr = 0; //name address of the max value.
    bool flag = false;
    //According to the input, LS_D mode will output all files ordered by modifed time
    if(op == LS_D){
	    printf("===sort by modified time===\n");
        for(int i = fs->SUPERBLOCK_SIZE; i < fs->FILE_BASE_ADDRESS; i += 32){
            for(int j = fs->SUPERBLOCK_SIZE; j < fs->FILE_BASE_ADDRESS; j += 32){
                if(fs->volume[j] != 0){
                    time_num = u8tou32(fs, j + 28);
//                    printf("%d\n", time_num);
                    if(time_num <= max_num) continue;
                    index = get_index(fs, j);
                    //Check the used array to check whether this index is already be used.
                    if(is_used(fs, index)){
                        max_index = index;
                        max_addr = j;
                        max_num = time_num;
                        flag = true;
                    }
                    else continue;
                }
                // = 0: This FCB entry is empty
                else continue;
            }
           //If no information is updated: We are Done
           if(flag){
                //Update the used array
                update_used(fs, max_index);
                //Output the filename
                for(int k = 0; k < 20; ++k){
                    if(fs->volume[max_addr + k] != 0){
                        printf("%c", fs->volume[max_addr + k]);
                    }
                    else{
                        printf("\n");
                        break;
                    }
                }
                //Reset the maximum value;
                max_num = 0;
                flag = false;
            }
            else break;

        }
	}
	else if(op == LS_S){
        //LS_S mode will output all files ordered by file size
        printf("===sort by file size===\n");
        for(int i = fs->SUPERBLOCK_SIZE; i < fs->FILE_BASE_ADDRESS; i += 32){
            //This variable is the modified time of the biggest size file.
            u32 modified_time = 0;
            for(int j = fs->SUPERBLOCK_SIZE; j < fs->FILE_BASE_ADDRESS; j += 32){
                if(fs->volume[j] != 0){
                    size_num = u8tou32(fs, j + 24);
                    time_num = u8tou32(fs, j + 28);
                    if(size_num < max_num) continue;
                    else if(size_num > max_num){
                        index = get_index(fs, j);
                        //Check the used array to check whether this index is already be used.
                        if(is_used(fs, index)){
                            modified_time = time_num;
                            max_index = index;
                            max_addr = j;
                            max_num = size_num;
                            flag = true;
                        }
                        else continue;
                    }
                }
                    // = 0: This FCB entry is empty
                else continue;
            }
            if(flag){
                //Update the used array
                update_used(fs, max_index);
                //Output the filename
                for(int k = 0; k < 20; ++k){
                    if(fs->volume[max_addr + k] != 0){
                        printf("%c", fs->volume[max_addr + k]);
                    }
                    else{
                        printf("  %u\n", u8tou32(fs, max_addr + 24));
                        break;
                    }
                }
                //Reset the maximum value;
                max_num = 0;
                flag = false;
            }
            else break;


        }
	}
}

// This function will remove the file given the file name.
__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
    //Find the file's FCB first:
    u32 file_index = find_file(fs, s);
    //We free the space first:
    u32 start_addr = u8tou32(fs, file_index + 20);
    u32 file_size = u8tou32(fs, file_index + 24);
    for(int i = 0; i < file_size; ++i){
        fs->volume[start_addr + i] = 0;
    }
    //Modify the bit-map --- change all the following bits to 1
    int block = file_size / 32;
    if(file_size % 32 != 0) ++block;
    updateSuperBlock(fs, start_addr + 32 * block, 0, 1);
    //This operation will compact the space and modify the bit map.
    compaction(fs, start_addr + block * 32);
    //Then we compact the FCB table, which also override the deleted file's FCB:
    for(u32 i = file_index; i < fs->FILE_BASE_ADDRESS - 64; i += 32){
        for(u32 j = 0; j < 32; ++j){
            fs->volume[i + j] = fs->volume[i + 32 + j];
        }
    }

}
